#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: P = M * N.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////

    __global__ void
matrixMul(
    float* P, const float* M, const float* N,
    const int Mh, const int Mw, const int Nw,
    const int block_size)
{
        const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    float Psub = 0;
    int i = 0, indexM = 0, indexN = 0, indexP = 0;

    // ===================================================================
    // Code Segment 5
    // Determine the output index of each thread.
    // Compute the dot product of one row of M and one column of N
    // for each thread.
    // Write the computed value to matrix P at the correct index.
    // ===================================================================
    int indX = bx * blockDim.x + tx;
    int indY = by * blockDim.y + ty;

    int nElementosFila = gridDim.x * blockDim.x;

    /*if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 1 && blockIdx.y == 0)
	printf("indX: %d, indY: %d, nElementosFila: %d\n", indX, indY, nElementosFila);
    */
    for (i = 0; i < Mw; ++i)
    {
//Psub += M[indX][i] * N[i][indY];

        indexM = (indX * nElementosFila) + i;
        indexN = indY + (nElementosFila * i);

        Psub += M[indexM] * N[indexN];

        indexP = (indX * gridDim.x * blockDim.x) + indY;

        /*if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 1 && blockIdx.y == 0)
	    //printf("Indexes -> %d %d: %d - %d\n", blockIdx.x * block_size + i, blockIdx.y * block_size + i, indexM, indexN);
	    printf("bx: %d, by: %d, tx: %d, ty: %d, indX: %d, intY: %d, indexP: %d, indexM: %d, indexN: %d, M[indexM]: %f, N[indexN]: %f, Psub: %f\n", bx, by, tx, ty, indX, indY, indexP, indexM, indexN, M[indexM], N[indexN], Psub);*/
    }


    P[indexP] = Psub;

	//printf("Result -> Psub: %f -> P[indexP]: %f\n\n\n", P[indexP]);

    // End of Code Segment 5 ============================================
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_


